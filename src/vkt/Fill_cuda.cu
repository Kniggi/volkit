#include "hip/hip_runtime.h"
#include <cstddef>
#include <cstdint>

#include "Fill_cuda.hpp"
#include "macros.hpp"

namespace vkt
{
    __constant__ uint8_t deviceMappedVoxel[StructuredVolume::GetMaxBytesPerVoxel()];

    __global__ void Fill_kernel(
            uint8_t* data,
            vec3i dims,
            uint16_t bytesPerVoxel,
            vec3i first,
            vec3i last
            )
    {
        int nx = last.x - first.x;
        int ny = last.y - first.y;
        int nz = last.z - first.z;

        int x = (blockIdx.x * blockDim.x + threadIdx.x) - first.x;
        int y = (blockIdx.y * blockDim.y + threadIdx.y) - first.y;
        int z = (blockIdx.z * blockDim.z + threadIdx.z) - first.z;

        if (x < nx && y < ny && z < nz)
        {
            std::size_t linearIndex = z * static_cast<std::size_t>(dims.x) * dims.y
                                    + y * dims.x
                                    + x;
            linearIndex *= bytesPerVoxel;

            for (uint16_t i = 0; i < bytesPerVoxel; ++i)
                data[linearIndex + i] = deviceMappedVoxel[i];
        }
    }

    void FillRange_cuda(StructuredVolume& volume, vec3i first, vec3i last, float value)
    {
        uint8_t mappedVoxel[StructuredVolume::GetMaxBytesPerVoxel()];
        volume.mapVoxel(mappedVoxel, value);

        VKT_CUDA_SAFE_CALL__(hipMemcpyToSymbol(HIP_SYMBOL(
                deviceMappedVoxel),
                mappedVoxel,
                StructuredVolume::GetMaxBytesPerVoxel(),
                0,
                hipMemcpyHostToDevice
                ));

        unsigned nx = last.x - first.x;
        unsigned ny = last.y - first.y;
        unsigned nz = last.z - first.z;

        dim3 blockSize(8, 8, 8);
        dim3 gridSize(
                div_up(nx, blockSize.x),
                div_up(ny, blockSize.y),
                div_up(nz, blockSize.z)
                );

        Fill_kernel<<<gridSize, blockSize>>>(
                volume.getData(),
                volume.getDims(),
                volume.getBytesPerVoxel(),
                first,
                last
                );
    }
} // vkt
